#include "hip/hip_runtime.h"
/*

This is an example for making tiling/blocking in openMP.
The calculation purpose is to reach a lot of memory from a single kernel.

Cpu ram-> gpu vram -> compute -> gpu vram- > cpu ram
*/

//stopper functions defined here, you may choose not to use them with -D NO_TIME

void Spawn_stopper(char *name);
double Kill_stopper();

#ifndef L3D
#define L1D
#endif

#include "stdlib.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include "omp.h"

//This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//#define THREADLIMIT 256

/* constans méretek */
/*
#define X 800
#define Y 800
#define Z 800
*/

const long sizex = 880;
const long sizey = 880;
const long sizez = 880;

struct dataobj{void *data;};


__constant__ float r8;
__constant__ float r9;



#ifdef L1D
  #define Ddim(arr,x,y,z) arr[(z) + (y)*sizez + (x)*sizez * sizey]

#elif L3D
  #define Ddim(arr,x,y,z) arr ## _[x][y][z]
#else
  #pragma message "Dimension was not defined use one of L1D or L3D " 
#endif

#if defined(F800_2)
#define window_size 2
#define KERNEL_WINDOW2(out,x,y,z) float r10 = 1.0F/(Ddim(data1,x + 2, y + 2, z + 2)*Ddim(data1,x + 2, y + 2, z + 2)); \
Ddim(out, x + 2, y + 2, z + 2) = (r9*Ddim(data2, x + 1, y + 1, z + 1)*Ddim(data3, x + 2, y + 2, z + 2) + r10*(-r8*(-2.0F*Ddim(data3, x + 2, y + 2, z + 2)) - r8*Ddim(data4, x + 2, y + 2, z + 2)) + 4.44444e-3F*(Ddim(data3, x + 1, y + 2, z + 2) + Ddim(data3, x + 2, y + 1, z + 2) + Ddim(data3, x + 2, y + 2, z + 1) + Ddim(data3, x + 2, y + 2, z + 3) + Ddim(data3, x + 2, y + 3, z + 2) + Ddim(data3, x + 3, y + 2, z + 2)) - 2.66666673e-2F*Ddim(data3, x + 2, y + 2, z + 2))/(r8*r10 + r9*Ddim(data2, x + 1, y + 1, z + 1));
#elif defined(F800_4)
#define window_size 4
#define KERNEL_WINDOW2(out,x,y,z) float r10 = 1.0F/(Ddim(data1,x + 4, y + 4, z + 4)*Ddim(data1,x + 4, y + 4, z + 4)); \
Ddim(out, x + 4, y + 4, z + 4) = (r9*Ddim(data2, x + 1, y + 1, z + 1)*Ddim(data3, x + 4, y + 4, z + 4) + r10*(-r8*(-2.0F*Ddim(data3, x + 4, y + 4, z + 4)) - r8*Ddim(data4, x + 4, y + 4, z + 4)) + 3.70370379e-4F*(-Ddim(data3, x + 2, y + 4, z + 4) - Ddim(data3, x + 4, y + 2, z + 4) - Ddim(data3, x + 4, y + 4, z + 2) - Ddim(data3, x + 4, y + 4, z + 6) - Ddim(data3, x + 4, y + 6, z + 4) - Ddim(data3, x + 6, y + 4, z + 4)) + 5.92592607e-3F*(Ddim(data3, x + 3, y + 4, z + 4) + Ddim(data3, x + 4, y + 3, z + 4) + Ddim(data3, x + 4, y + 4, z + 3) + Ddim(data3, x + 4, y + 4, z + 5) + Ddim(data3, x + 4, y + 5, z + 4) + Ddim(data3, x + 5, y + 4, z + 4)) - 3.33333341e-2F*Ddim(data3, x + 4, y + 4, z + 4))/(r8*r10 + r9*Ddim(data2, x + 1, y + 1, z + 1));
#elif defined(F800_8)
#define window_size 8
#define KERNEL_WINDOW2(out,x,y,z) float r10 = 1.0F/(Ddim(data1,x + 8, y + 8, z + 8)*Ddim(data1,x + 8, y + 8, z + 8)); \
Ddim(out, x + 8, y + 8, z + 8) = (r9*Ddim(data2, x + 1, y + 1, z + 1)*Ddim(data3, x + 8, y + 8, z + 8) + r10*(-r8*(-2.0F*Ddim(data3, x + 8, y + 8, z + 8)) - r8*Ddim(data4, x + 8, y + 8, z + 8)) + 7.93650813e-6F*(-Ddim(data3, x + 4, y + 8, z + 8) - Ddim(data3, x + 8, y + 4, z + 8) - Ddim(data3, x + 8, y + 8, z + 4) - Ddim(data3, x + 8, y + 8, z + 12) - Ddim(data3, x + 8, y + 12, z + 8) - Ddim(data3, x + 12, y + 8, z + 8)) + 1.12874782e-4F*(Ddim(data3, x + 5, y + 8, z + 8) + Ddim(data3, x + 8, y + 5, z + 8) + Ddim(data3, x + 8, y + 8, z + 5) + Ddim(data3, x + 8, y + 8, z + 11) + Ddim(data3, x + 8, y + 11, z + 8) + Ddim(data3, x + 11, y + 8, z + 8)) + 8.8888891e-4F*(-Ddim(data3, x + 6, y + 8, z + 8) - Ddim(data3, x + 8, y + 6, z + 8) - Ddim(data3, x + 8, y + 8, z + 6) - Ddim(data3, x + 8, y + 8, z + 10) - Ddim(data3, x + 8, y + 10, z + 8) - Ddim(data3, x + 10, y + 8, z + 8)) + 7.11111128e-3F*(Ddim(data3, x + 7, y + 8, z + 8) + Ddim(data3, x + 8, y + 7, z + 8) + Ddim(data3, x + 8, y + 8, z + 7) + Ddim(data3, x + 8, y + 8, z + 9) + Ddim(data3, x + 8, y + 9, z + 8) + Ddim(data3, x + 9, y + 8, z + 8)) - 3.79629639e-2F*Ddim(data3, x + 8, y + 8, z + 8))/(r8*r10 + r9*Ddim(data2, x + 1, y + 1, z + 1));
#elif defined(F800_16)
#define window_size 16
#define KERNEL_WINDOW2(out,x,y,z) float r10 = 1.0F/(Ddim(data1,x + 16, y + 16, z + 16)*Ddim(data1,x + 16, y + 16, z + 16)); \
Ddim(out, x + 16, y + 16, z + 16) = (r9*Ddim(data2, x + 1, y + 1, z + 1)*Ddim(data3, x + 16, y + 16, z + 16) + r10*(-r8*(-2.0F*Ddim(data3, x + 16, y + 16, z + 16)) - r8*Ddim(data4, x + 16, y + 16, z + 16)) + 1.07916777e-8F*(-Ddim(data3, x + 8, y + 16, z + 16) - Ddim(data3, x + 16, y + 8, z + 16) - Ddim(data3, x + 16, y + 16, z + 8) - Ddim(data3, x + 16, y + 16, z + 24) - Ddim(data3, x + 16, y + 24, z + 16) - Ddim(data3, x + 24, y + 16, z + 16)) + 2.25524041e-7F*(Ddim(data3, x + 9, y + 16, z + 16) + Ddim(data3, x + 16, y + 9, z + 16) + Ddim(data3, x + 16, y + 16, z + 9) + Ddim(data3, x + 16, y + 16, z + 23) + Ddim(data3, x + 16, y + 23, z + 16) + Ddim(data3, x + 23, y + 16, z + 16)) + 2.30222458e-6F*(-Ddim(data3, x + 10, y + 16, z + 16) - Ddim(data3, x + 16, y + 10, z + 16) - Ddim(data3, x + 16, y + 16, z + 10) - Ddim(data3, x + 16, y + 16, z + 22) - Ddim(data3, x + 16, y + 22, z + 16) - Ddim(data3, x + 22, y + 16, z + 16)) + 1.54709492e-5F*(Ddim(data3, x + 11, y + 16, z + 16) + Ddim(data3, x + 16, y + 11, z + 16) + Ddim(data3, x + 16, y + 16, z + 11) + Ddim(data3, x + 16, y + 16, z + 21) + Ddim(data3, x + 16, y + 21, z + 16) + Ddim(data3, x + 21, y + 16, z + 16)) + 7.85634138e-5F*(-Ddim(data3, x + 12, y + 16, z + 16) - Ddim(data3, x + 16, y + 12, z + 16) - Ddim(data3, x + 16, y + 16, z + 12) - Ddim(data3, x + 16, y + 16, z + 20) - Ddim(data3, x + 16, y + 20, z + 16) - Ddim(data3, x + 20, y + 16, z + 16)) + 3.35203899e-4F*(Ddim(data3, x + 13, y + 16, z + 16) + Ddim(data3, x + 16, y + 13, z + 16) + Ddim(data3, x + 16, y + 16, z + 13) + Ddim(data3, x + 16, y + 16, z + 19) + Ddim(data3, x + 16, y + 19, z + 16) + Ddim(data3, x + 19, y + 16, z + 16)) + 1.38271608e-3F*(-Ddim(data3, x + 14, y + 16, z + 16) - Ddim(data3, x + 16, y + 14, z + 16) - Ddim(data3, x + 16, y + 16, z + 14) - Ddim(data3, x + 16, y + 16, z + 18) - Ddim(data3, x + 16, y + 18, z + 16) - Ddim(data3, x + 18, y + 16, z + 16)) + 7.90123476e-3F*(Ddim(data3, x + 15, y + 16, z + 16) + Ddim(data3, x + 16, y + 15, z + 16) + Ddim(data3, x + 16, y + 16, z + 15) + Ddim(data3, x + 16, y + 16, z + 17) + Ddim(data3, x + 16, y + 17, z + 16) + Ddim(data3, x + 17, y + 16, z + 16)) - 4.07312557e-2F*Ddim(data3, x + 16, y + 16, z + 16))/(r8*r10 + r9*Ddim(data2, x + 1, y + 1, z + 1));
#else
#error "invalid space order"
#endif









__global__ void cuda_section0(float * data,float * data1,float * data2,float * data3,float * data4,float * out){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blocksize_z + 0;
  int y = threadIdx.y + blockIdx.y * blocksize_y + 0;
  int x = threadIdx.z + blockIdx.z * blocksize_x + 0; 
	//if (x <= x_M && y <= y_M && z <= z_M){
		if (x < 880 - 2*window_size && y < 880 - 2*window_size && z < 880 - 2*window_size){		
			KERNEL_WINDOW2(out,x,y,z);
	}
}
__global__ void cuda_init(float * data,float * data1,float * data2,float * data3,float * data4,float * out){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blocksize_z + 0;
  int y = threadIdx.y + blockIdx.y * blocksize_y + 0;
  int x = threadIdx.z + blockIdx.z * blocksize_x + 0;
		//kernel start
		
		if (x < 880 && y < 880 && z < 880){		
		Ddim(data,x,y,z) = 1.0;
		Ddim(data1,x,y,z) = 1.0;
		Ddim(data2,x,y,z) = 1.0;
		Ddim(data3,x,y,z) = 1.0;
		Ddim(data4,x,y,z) = 1.0;
		Ddim(out,x,y,z) = 0.0;
		//Ddim(out2,x,y,z) = 0.0;
	}
}

extern "C" void cuda_enter_data(float** device_ptr,float* data,size_t size)
{
	cudaCheck(hipMalloc(device_ptr, size*sizeof(float)));
	cudaCheck(hipMemcpy(*device_ptr, data,size*sizeof(float), hipMemcpyHostToDevice));
}
extern "C" void cuda_exit_data(float* device_ptr){
	cudaCheck(hipFree(device_ptr));
}
extern "C" void cuda_update_data_from(float* device_ptr,float* data,size_t size){
	//print0<<<1,1>>>(device_ptr);
	cudaCheck(hipMemcpy(data,device_ptr,size*sizeof(float), hipMemcpyDeviceToHost));
	cudaCheck(hipFree(device_ptr));
	//printf("cpu %f,\n",(data)[1518766805]);
}


int main(int argc, char **argv)
{


#ifdef L1D
  printf("compiled with 1 dimensional arrays.\n");
#elif L3D
  printf("compiled with 3 dimensional arrays.\n");
#else
  #pragma message "Dimension was not defined use one of L1D or L3D " 
#endif


#if defined(FORBLOCKED) || defined(FORBLOCKTILLED) || defined(FORTILLED)
printf("block size: %d,%d,%d \n", blocksize_x, blocksize_y, blocksize_z);
#endif
const size_t meret = sizex * sizey * sizez;
printf("memory size needed: %lu , %f Gb \n", meret, 7*meret * sizeof(float) / 1e9f);


float * cpu_out = (float *)malloc(meret * sizeof(float));
float * cpu_out2 = (float *)malloc(meret * sizeof(float));

float * out;
float * out2;

float * data;
float * data1;
float * data2;
float * data3;
float * data4;

cudaCheck(hipMalloc(&data, meret*sizeof(float)));
cudaCheck(hipMalloc(&data1, meret*sizeof(float)));
cudaCheck(hipMalloc(&data2, meret*sizeof(float)));
cudaCheck(hipMalloc(&data3, meret*sizeof(float)));
cudaCheck(hipMalloc(&data4, meret*sizeof(float)));
cudaCheck(hipMalloc(&out, meret*sizeof(float)));
cudaCheck(hipMalloc(&out2, meret*sizeof(float))); //nem fontos amúgy
printf("gpu : %p\n",data);

float dt = 1.816000;
float r8_ = 1.0F/(dt*dt);
float r9_ = 1.0F/dt;
cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(r8),&r8_, sizeof(float)));
cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(r9),&r9_, sizeof(float)));

//#pragma omp target data map(alloc : data[0:meret]) \
//			map(alloc : data1[0:meret]) \
//			map(alloc : data2[0:meret]) \
//			map(alloc : data3[0:meret]) \
//			map(alloc : data4[0:meret]) \
//			map(from  : out[0:meret])  \
//			map(from  : out2[0:meret])
{


{
	//init aka zeroing
	dim3 threads(blocksize_z,blocksize_y,blocksize_x);
	dim3 blocks((880)/blocksize_z+1,(880)/blocksize_y+1,(880)/blocksize_x+1);
	cuda_init<<<blocks,threads>>>(data,data1,data2,data3,data4,out);
	cudaCheck(hipPeekAtLastError());
	cudaCheck(hipDeviceSynchronize());
}


dim3 threads(blocksize_z,blocksize_y,blocksize_x);
dim3 blocks((880 - 2*window_size)/blocksize_z+1,(880 - 2*window_size)/blocksize_y+1,(880 - 2*window_size)/blocksize_x+1);


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
for (int i = 0; i < 21; i++) {
	if (i == 11) 
	{
		cudaCheck(hipDeviceSynchronize());
		Spawn_stopper("Kernel 0");	
	}
	cuda_section0<<<blocks,threads>>>(data,data1,data2,data3,data4,out);
	cudaCheck(hipPeekAtLastError());
	
}
cudaCheck(hipPeekAtLastError());
cudaCheck(hipDeviceSynchronize());
Kill_stopper();

}
//printf("gpu : %p\n",data);
//printf("gpu : %p\n",data1);

cudaCheck(hipFree(data));
cudaCheck(hipFree(data1));
cudaCheck(hipFree(data2));
cudaCheck(hipFree(data3));
cudaCheck(hipFree(data4));
cudaCheck(hipMemcpy(cpu_out,out,meret*sizeof(float), hipMemcpyDeviceToHost));
cudaCheck(hipFree(out));
printf("cudafree done.\n");
//cuda_update_data_from(out,cpu_out,meret);
//cuda_update_data_from(out2,cpu_out2,meret);

///////////////////////////////////////////////////////////////////////////////////////////////////////////////
#ifdef EXPORTDATA
{
	FILE *write_ptr;
	write_ptr = fopen("./out.bin","wb");  // w for write, b for binary
	size_t arrsize = meret * sizeof(float);
	fwrite((char*)cpu_out,arrsize,1,write_ptr); // write bytes from our buffer

}
#endif

free(cpu_out);
free(cpu_out2);

printf("Done.\n");

return 0;
}

//END of example

//If you dont want to compile this code use -D NO_TIME in compile option.

//This code is for the stopper not part of the example
//Do not change this code
//The example code is up

#ifdef NO_TIME
void Spawn_stopper(char* name){};
double Kill_stopper(){};
#else
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>


// proper definition of the stopper
struct Stopper
{
	/* data */
	struct Stopper* parent; //am i a child
	double start,end;	   //preaty self explain
	char name[256];		  //20 caracter for a name is enought //update it is few week later and 256 because turns out it isn't enought, this took like 3 painfull hours to find out.
	
};
//typedef struct Stopper Stopper();

double Totaltime=0;
int Sum_Stopper=0;
// Stoppers follow LIFO structure
struct Stopper * Stopper_root=NULL; //root of the timers

//get the curent time
//also it does magic because i dont remember what it does actualy
double op_timer_core() {
struct timeval t;
gettimeofday(&t, (struct timezone *)0);
return t.tv_sec + t.tv_usec * 1.0e-6;
}




void Spawn_stopper(char* name)
{
	  struct Stopper * stopper= (Stopper *)malloc(sizeof(struct Stopper));
	  // printf("%s ID:%d started->\n",name,Sum_Stopper);
	  strcpy(stopper->name,name);
	  stopper->start=op_timer_core();
	  stopper->parent=Stopper_root;
	  Stopper_root=stopper;
	  Sum_Stopper++;
}
double Kill_stopper()
{
  double ret=0;
  if(Stopper_root==NULL)
	  return ret;
  Stopper_root->end=op_timer_core();
  ret=Stopper_root->end-=Stopper_root->start;
  printf("%s %f sec.\n",Stopper_root->name,Stopper_root->end);
  struct Stopper* t=Stopper_root;//free the stopper
  Stopper_root=Stopper_root->parent;
  free(t);
  return ret;

}
#endif

