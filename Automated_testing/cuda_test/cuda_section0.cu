#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_section0.h"

//#pragma omp target teams distribute parallel for collapse(3)
//for (int x = x_m; x <= x_M; x += 1)
//{
//  for (int y = y_m; y <= y_M; y += 1)
//  {
//    for (int z = z_m; z <= z_M; z += 1)
//    {
//      float r10 = 1.0F/(vpL0(x + 16, y + 16, z + 16)*vpL0(x + 16, y + 16, z + 16));
//      uL0(t2, x + 16, y + 16, z + 16) = (r9*dampL0(x + 1, y + 1, z + 1)*uL0(t0, x + 16, y + 16, z + 16) + r10*(-r8*(-2.0F*uL0(t0, x + 16, y + 16, z + 16)) - r8*uL0(t1, x + 16, y + 16, z + 16)) + 1.07916777e-8F*(-uL0(t0, x + 8, y + 16, z + 16) - uL0(t0, x + 16, y + 8, z + 16) - uL0(t0, x + 16, y + 16, z + 8) - uL0(t0, x + 16, y + 16, z + 24) - uL0(t0, x + 16, y + 24, z + 16) - uL0(t0, x + 24, y + 16, z + 16)) + 2.25524041e-7F*(uL0(t0, x + 9, y + 16, z + 16) + uL0(t0, x + 16, y + 9, z + 16) + uL0(t0, x + 16, y + 16, z + 9) + uL0(t0, x + 16, y + 16, z + 23) + uL0(t0, x + 16, y + 23, z + 16) + uL0(t0, x + 23, y + 16, z + 16)) + 2.30222458e-6F*(-uL0(t0, x + 10, y + 16, z + 16) - uL0(t0, x + 16, y + 10, z + 16) - uL0(t0, x + 16, y + 16, z + 10) - uL0(t0, x + 16, y + 16, z + 22) - uL0(t0, x + 16, y + 22, z + 16) - uL0(t0, x + 22, y + 16, z + 16)) + 1.54709492e-5F*(uL0(t0, x + 11, y + 16, z + 16) + uL0(t0, x + 16, y + 11, z + 16) + uL0(t0, x + 16, y + 16, z + 11) + uL0(t0, x + 16, y + 16, z + 21) + uL0(t0, x + 16, y + 21, z + 16) + uL0(t0, x + 21, y + 16, z + 16)) + 7.85634138e-5F*(-uL0(t0, x + 12, y + 16, z + 16) - uL0(t0, x + 16, y + 12, z + 16) - uL0(t0, x + 16, y + 16, z + 12) - uL0(t0, x + 16, y + 16, z + 20) - uL0(t0, x + 16, y + 20, z + 16) - uL0(t0, x + 20, y + 16, z + 16)) + 3.35203899e-4F*(uL0(t0, x + 13, y + 16, z + 16) + uL0(t0, x + 16, y + 13, z + 16) + uL0(t0, x + 16, y + 16, z + 13) + uL0(t0, x + 16, y + 16, z + 19) + uL0(t0, x + 16, y + 19, z + 16) + uL0(t0, x + 19, y + 16, z + 16)) + 1.38271608e-3F*(-uL0(t0, x + 14, y + 16, z + 16) - uL0(t0, x + 16, y + 14, z + 16) - uL0(t0, x + 16, y + 16, z + 14) - uL0(t0, x + 16, y + 16, z + 18) - uL0(t0, x + 16, y + 18, z + 16) - uL0(t0, x + 18, y + 16, z + 16)) + 7.90123476e-3F*(uL0(t0, x + 15, y + 16, z + 16) + uL0(t0, x + 16, y + 15, z + 16) + uL0(t0, x + 16, y + 16, z + 15) + uL0(t0, x + 16, y + 16, z + 17) + uL0(t0, x + 16, y + 17, z + 16) + uL0(t0, x + 17, y + 16, z + 16)) - 4.07312557e-2F*uL0(t0, x + 16, y + 16, z + 16))/(r8*r10 + r9*dampL0(x + 1, y + 1, z + 1));
//    }
//  }
//}

//This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


//#ifndef blocksize_x
//#define blocksize_x 8
//#endif
//#ifndef blocksize_y
//#define blocksize_y 4
//#endif
//#ifndef blocksize_z
//#define blocksize_z 32
//#endif

#define Ddim(arr,x,y,z) arr[(z) + (y)*SIZEZ + (x)*SIZEZ * SIZEY]

//const int sizex = SIZEX;
//const int sizey = SIZEY;
//const int sizez = SIZEZ;

#define uL0(t, x, y, z) u[(t)*x_stride0 + (x)*y_stride0 + (y)*z_stride0 + (z)]
#define dampL0(x, y, z) damp[(x)*y_stride1 + (y)*z_stride1 + (z)]
#define vpL0(x, y, z) vp[(x)*y_stride0 + (y)*z_stride0 + (z)]

__constant__ float r8;
__constant__ float r9;

__constant__ long x_stride0;
__constant__ long y_stride0;
__constant__ long z_stride0;
__constant__ long y_stride1;
__constant__ long z_stride1;
__constant__ long p_rec_stride0;
__constant__ long d_stride0;
__constant__ long p_src_stride0;


__global__ void cuda_section0(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blocksize_z + z_m;
  int y = threadIdx.y + blockIdx.y * blocksize_y + y_m;
  int x = threadIdx.z + blockIdx.z * blocksize_x + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){
		//printf("section kernel v: %f\n",vp[0]);
		//printf("%f,",vp[(x)*y_stride0 + (y)*z_stride0 + (z)]);


		#include "../dimension_src/func_800_2.h"
		#include "../dimension_src/func_800_4.h"
		#include "../dimension_src/func_800_8.h"
		#include "../dimension_src/func_800_16.h"

		//float r10 = 1.0F/(vpL0(x + 16, y + 16, z + 16)*vpL0(x + 16, y + 16, z + 16));
    	//uL0(t2, x + 16, y + 16, z + 16) = (r9*dampL0(x + 1, y + 1, z + 1)*uL0(t0, x + 16, y + 16, z + 16) + r10*(-r8*(-2.0F*uL0(t0, x + 16, y + 16, z + 16)) - r8*uL0(t1, x + 16, y + 16, z + 16)) + 1.07916777e-8F*(-uL0(t0, x + 8, y + 16, z + 16) - uL0(t0, x + 16, y + 8, z + 16) - uL0(t0, x + 16, y + 16, z + 8) - uL0(t0, x + 16, y + 16, z + 24) - uL0(t0, x + 16, y + 24, z + 16) - uL0(t0, x + 24, y + 16, z + 16)) + 2.25524041e-7F*(uL0(t0, x + 9, y + 16, z + 16) + uL0(t0, x + 16, y + 9, z + 16) + uL0(t0, x + 16, y + 16, z + 9) + uL0(t0, x + 16, y + 16, z + 23) + uL0(t0, x + 16, y + 23, z + 16) + uL0(t0, x + 23, y + 16, z + 16)) + 2.30222458e-6F*(-uL0(t0, x + 10, y + 16, z + 16) - uL0(t0, x + 16, y + 10, z + 16) - uL0(t0, x + 16, y + 16, z + 10) - uL0(t0, x + 16, y + 16, z + 22) - uL0(t0, x + 16, y + 22, z + 16) - uL0(t0, x + 22, y + 16, z + 16)) + 1.54709492e-5F*(uL0(t0, x + 11, y + 16, z + 16) + uL0(t0, x + 16, y + 11, z + 16) + uL0(t0, x + 16, y + 16, z + 11) + uL0(t0, x + 16, y + 16, z + 21) + uL0(t0, x + 16, y + 21, z + 16) + uL0(t0, x + 21, y + 16, z + 16)) + 7.85634138e-5F*(-uL0(t0, x + 12, y + 16, z + 16) - uL0(t0, x + 16, y + 12, z + 16) - uL0(t0, x + 16, y + 16, z + 12) - uL0(t0, x + 16, y + 16, z + 20) - uL0(t0, x + 16, y + 20, z + 16) - uL0(t0, x + 20, y + 16, z + 16)) + 3.35203899e-4F*(uL0(t0, x + 13, y + 16, z + 16) + uL0(t0, x + 16, y + 13, z + 16) + uL0(t0, x + 16, y + 16, z + 13) + uL0(t0, x + 16, y + 16, z + 19) + uL0(t0, x + 16, y + 19, z + 16) + uL0(t0, x + 19, y + 16, z + 16)) + 1.38271608e-3F*(-uL0(t0, x + 14, y + 16, z + 16) - uL0(t0, x + 16, y + 14, z + 16) - uL0(t0, x + 16, y + 16, z + 14) - uL0(t0, x + 16, y + 16, z + 18) - uL0(t0, x + 16, y + 18, z + 16) - uL0(t0, x + 18, y + 16, z + 16)) + 7.90123476e-3F*(uL0(t0, x + 15, y + 16, z + 16) + uL0(t0, x + 16, y + 15, z + 16) + uL0(t0, x + 16, y + 16, z + 15) + uL0(t0, x + 16, y + 16, z + 17) + uL0(t0, x + 16, y + 17, z + 16) + uL0(t0, x + 17, y + 16, z + 16)) - 4.07312557e-2F*uL0(t0, x + 16, y + 16, z + 16))/(r8*r10 + r9*dampL0(x + 1, y + 1, z + 1));
	  
		
		//printf("uL0(t2, x + 16, y + 16, z + 16) = %f \n",uL0(t2, x + 16, y + 16, z + 16));
	}

	//if(x==0 && y==0 && z==0)
	//{
		//printf("uL0(t2, x + 16, y + 16, z + 16) = %f\n",uL0(t2, x + 16, y + 16, z + 16));
	//}

}
/*
__global__ void print(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blockDim.x + z_m;
  int y = threadIdx.y + blockIdx.y * blockDim.y + y_m;
  int x = threadIdx.z + blockIdx.z * blockDim.z + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){
		printf("kernel:\nv %f,",vpL0(x, y, z));
		printf("u %f,",uL0(t2, x, y, z));
		printf("damp %f\n",dampL0(x, y, z));
	}
}
*/

__global__ void print0(float *test){
	printf("kernel: test %f\n",test[1530428310]);
}

__global__ void print1(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");
	
	int z = threadIdx.x + blockIdx.x * blocksize_z + z_m;
	int y = threadIdx.y + blockIdx.y * blocksize_y + y_m;
	int x = threadIdx.z + blockIdx.z * blocksize_x + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){
		printf("idx: %lu ,u_%f, trick %f\n",
		((t2)*x_stride0 + (x + 16)*y_stride0 + (y + 16)*z_stride0 + (z + 16)),
    	uL0(t2, x + 16, y + 16, z + 16),u[1530428310]
	);
	}

}



__global__ void cuda_test(){
	printf("section0 gpu\n");
}
/*
extern "C" void kernel_section0()
{
	printf("asdasd\n");
	cuda_test<<<2,2>>>();
	hipDeviceSynchronize();
}
*/


extern "C" void kernel_vars(const float dt,const long x_stride0_,const long y_stride0_,const long z_stride0_,const long y_stride1_,const long z_stride1_,const long p_rec_stride0_,const long d_stride0_,const long p_src_stride0_)
{
	//TODO errors
	float r8_ = 1.0F/(dt*dt);
	float r9_ = 1.0F/dt;
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(r8),&r8_, sizeof(float)));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(r9),&r9_, sizeof(float)));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( x_stride0), &x_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( y_stride0), &y_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( z_stride0), &z_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( y_stride1), &y_stride1_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( z_stride1), &z_stride1_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( p_rec_stride0), &p_rec_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( d_stride0), &d_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( p_src_stride0), &p_src_stride0_, sizeof(long) ));
}



extern "C" void kernel_section0( const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m, int t0,int t1,int t2,float *vp,float *u,float *damp)
{

	dim3 threads(blocksize_z,blocksize_y,blocksize_x);
	dim3 blocks((x_M-x_m-1)/blocksize_z+1,(y_M-y_m-1)/blocksize_y+1,(z_M-z_m-1)/blocksize_x+1);
	
	//printf("u pointer gpu: %p\n",u);
	//printf("damp pointer gpu: %p\n",damp);
	//printf("vp pointer gpu: %p\n",vp);
	
	cuda_section0<<<blocks,threads>>>(x_M,x_m,y_M,y_m,z_M,z_m,t0,t1,t2,vp,u,damp);
	
	//print1<<<blocks,threads>>>(x_M,x_m,y_M,y_m,z_M,z_m,t0,t1,t2,vp,u,damp);
	//print0<<<1,1>>>(u);
	
	
	//hipDeviceSynchronize();
}


extern "C" void cuda_enter_data(float** device_ptr,float* data,size_t size)
{
	//printf("%f\n",data[0]);
	//printf("size: %lu\n",size);
	//getchar();
	cudaCheck(hipMalloc(device_ptr, size*sizeof(float)));
	cudaCheck(hipMemcpy(*device_ptr, data,size*sizeof(float), hipMemcpyHostToDevice));
	//dim3 threads(1,1,1);
	//dim3 blocks(1,1,1);
	//print0<<<1,1>>>(*device_ptr);
	//hipDeviceSynchronize();
	//printf("%f\n",(*device_ptr)[0]);
}
extern "C" void cuda_exit_data(float* device_ptr){
	cudaCheck(hipFree(device_ptr));
}
extern "C" void cuda_update_data_from(float* device_ptr,float* data,size_t size){
	//print0<<<1,1>>>(device_ptr);
	cudaCheck(hipMemcpy(data,device_ptr,size*sizeof(float), hipMemcpyDeviceToHost));
	cudaCheck(hipFree(device_ptr));
	//printf("cpu %f,\n",(data)[1518766805]);
}



/*

int main()
{
	kernel_section0(10,10,10,10,10,10,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0);



	return 0;
}

*/

