#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_section0.h"


//This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


//#ifndef blocksize_x
//#define blocksize_x 8
//#endif
//#ifndef blocksize_y
//#define blocksize_y 4
//#endif
//#ifndef blocksize_z
//#define blocksize_z 32
//#endif

#define Ddim(arr,x,y,z) arr[(z) + (y)*SIZEZ + (x)*SIZEZ * SIZEY]


#define uL0(t, x, y, z) u[(t)*x_stride0 + (x)*y_stride0 + (y)*z_stride0 + (z)]
#define dampL0(x, y, z) damp[(x)*y_stride1 + (y)*z_stride1 + (z)]
#define vpL0(x, y, z) vp[(x)*y_stride0 + (y)*z_stride0 + (z)]

__constant__ float r8;
__constant__ float r9;

__constant__ long x_stride0;
__constant__ long y_stride0;
__constant__ long z_stride0;
__constant__ long y_stride1;
__constant__ long z_stride1;
__constant__ long p_rec_stride0;
__constant__ long d_stride0;
__constant__ long p_src_stride0;


__global__ void cuda_section0(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blocksize_z + z_m;
  int y = threadIdx.y + blockIdx.y * blocksize_y + y_m;
  int x = threadIdx.z + blockIdx.z * blocksize_x + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){

		#include "dimension_src/func_800_2.h"
		#include "dimension_src/func_800_4.h"
		#include "dimension_src/func_800_8.h"
		#include "dimension_src/func_800_16.h"

	}
}
/*
__global__ void print(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blockDim.x + z_m;
  int y = threadIdx.y + blockIdx.y * blockDim.y + y_m;
  int x = threadIdx.z + blockIdx.z * blockDim.z + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){
		printf("kernel:\nv %f,",vpL0(x, y, z));
		printf("u %f,",uL0(t2, x, y, z));
		printf("damp %f\n",dampL0(x, y, z));
	}
}
*/

__global__ void print0(float *test){
	printf("kernel: test %f\n",test[1530428310]);
}

__global__ void print1(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");
	
	int z = threadIdx.x + blockIdx.x * blocksize_z + z_m;
	int y = threadIdx.y + blockIdx.y * blocksize_y + y_m;
	int x = threadIdx.z + blockIdx.z * blocksize_x + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){
		printf("idx: %lu ,u_%f, trick %f\n",
		((t2)*x_stride0 + (x + 16)*y_stride0 + (y + 16)*z_stride0 + (z + 16)),
    	uL0(t2, x + 16, y + 16, z + 16),u[1530428310]
	);
	}

}

__global__ void cuda_test(){
	printf("section0 gpu\n");
}


extern "C" void kernel_vars(const float dt,const long x_stride0_,const long y_stride0_,const long z_stride0_,const long y_stride1_,const long z_stride1_,const long p_rec_stride0_,const long d_stride0_,const long p_src_stride0_)
{
	//TODO errors
	float r8_ = 1.0F/(dt*dt);
	float r9_ = 1.0F/dt;
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(r8),&r8_, sizeof(float)));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(r9),&r9_, sizeof(float)));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( x_stride0), &x_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( y_stride0), &y_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( z_stride0), &z_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( y_stride1), &y_stride1_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( z_stride1), &z_stride1_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( p_rec_stride0), &p_rec_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( d_stride0), &d_stride0_, sizeof(long) ));
	cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL( p_src_stride0), &p_src_stride0_, sizeof(long) ));
}



extern "C" void kernel_section0( const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m, int t0,int t1,int t2,float *vp,float *u,float *damp)
{

	dim3 threads(blocksize_z,blocksize_y,blocksize_x);
	dim3 blocks((x_M-x_m-1)/blocksize_z+1,(y_M-y_m-1)/blocksize_y+1,(z_M-z_m-1)/blocksize_x+1);
	
	//printf("u pointer gpu: %p\n",u);
	//printf("damp pointer gpu: %p\n",damp);
	//printf("vp pointer gpu: %p\n",vp);
	
	cuda_section0<<<blocks,threads>>>(x_M,x_m,y_M,y_m,z_M,z_m,t0,t1,t2,vp,u,damp);
	
	//print1<<<blocks,threads>>>(x_M,x_m,y_M,y_m,z_M,z_m,t0,t1,t2,vp,u,damp);
	//print0<<<1,1>>>(u);
	
	
	//hipDeviceSynchronize();
}


extern "C" void cuda_enter_data(float** device_ptr,float* data,size_t size)
{
	cudaCheck(hipMalloc(device_ptr, size*sizeof(float)));
	cudaCheck(hipMemcpy(*device_ptr, data,size*sizeof(float), hipMemcpyHostToDevice));
}
extern "C" void cuda_exit_data(float* device_ptr){
	cudaCheck(hipFree(device_ptr));
}
extern "C" void cuda_update_data_from(float* device_ptr,float* data,size_t size){
	//print0<<<1,1>>>(device_ptr);
	cudaCheck(hipMemcpy(data,device_ptr,size*sizeof(float), hipMemcpyDeviceToHost));
	cudaCheck(hipFree(device_ptr));
	//printf("cpu %f,\n",(data)[1518766805]);
}


