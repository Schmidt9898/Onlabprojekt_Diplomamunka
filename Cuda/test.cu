#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<2,2>>>(); 
	hipDeviceSynchronize();
    return 0;
}