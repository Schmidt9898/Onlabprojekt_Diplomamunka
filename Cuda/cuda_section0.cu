#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_section0.h"

//#pragma omp target teams distribute parallel for collapse(3)
//for (int x = x_m; x <= x_M; x += 1)
//{
//  for (int y = y_m; y <= y_M; y += 1)
//  {
//    for (int z = z_m; z <= z_M; z += 1)
//    {
//      float r10 = 1.0F/(vpL0(x + 16, y + 16, z + 16)*vpL0(x + 16, y + 16, z + 16));
//      uL0(t2, x + 16, y + 16, z + 16) = (r9*dampL0(x + 1, y + 1, z + 1)*uL0(t0, x + 16, y + 16, z + 16) + r10*(-r8*(-2.0F*uL0(t0, x + 16, y + 16, z + 16)) - r8*uL0(t1, x + 16, y + 16, z + 16)) + 1.07916777e-8F*(-uL0(t0, x + 8, y + 16, z + 16) - uL0(t0, x + 16, y + 8, z + 16) - uL0(t0, x + 16, y + 16, z + 8) - uL0(t0, x + 16, y + 16, z + 24) - uL0(t0, x + 16, y + 24, z + 16) - uL0(t0, x + 24, y + 16, z + 16)) + 2.25524041e-7F*(uL0(t0, x + 9, y + 16, z + 16) + uL0(t0, x + 16, y + 9, z + 16) + uL0(t0, x + 16, y + 16, z + 9) + uL0(t0, x + 16, y + 16, z + 23) + uL0(t0, x + 16, y + 23, z + 16) + uL0(t0, x + 23, y + 16, z + 16)) + 2.30222458e-6F*(-uL0(t0, x + 10, y + 16, z + 16) - uL0(t0, x + 16, y + 10, z + 16) - uL0(t0, x + 16, y + 16, z + 10) - uL0(t0, x + 16, y + 16, z + 22) - uL0(t0, x + 16, y + 22, z + 16) - uL0(t0, x + 22, y + 16, z + 16)) + 1.54709492e-5F*(uL0(t0, x + 11, y + 16, z + 16) + uL0(t0, x + 16, y + 11, z + 16) + uL0(t0, x + 16, y + 16, z + 11) + uL0(t0, x + 16, y + 16, z + 21) + uL0(t0, x + 16, y + 21, z + 16) + uL0(t0, x + 21, y + 16, z + 16)) + 7.85634138e-5F*(-uL0(t0, x + 12, y + 16, z + 16) - uL0(t0, x + 16, y + 12, z + 16) - uL0(t0, x + 16, y + 16, z + 12) - uL0(t0, x + 16, y + 16, z + 20) - uL0(t0, x + 16, y + 20, z + 16) - uL0(t0, x + 20, y + 16, z + 16)) + 3.35203899e-4F*(uL0(t0, x + 13, y + 16, z + 16) + uL0(t0, x + 16, y + 13, z + 16) + uL0(t0, x + 16, y + 16, z + 13) + uL0(t0, x + 16, y + 16, z + 19) + uL0(t0, x + 16, y + 19, z + 16) + uL0(t0, x + 19, y + 16, z + 16)) + 1.38271608e-3F*(-uL0(t0, x + 14, y + 16, z + 16) - uL0(t0, x + 16, y + 14, z + 16) - uL0(t0, x + 16, y + 16, z + 14) - uL0(t0, x + 16, y + 16, z + 18) - uL0(t0, x + 16, y + 18, z + 16) - uL0(t0, x + 18, y + 16, z + 16)) + 7.90123476e-3F*(uL0(t0, x + 15, y + 16, z + 16) + uL0(t0, x + 16, y + 15, z + 16) + uL0(t0, x + 16, y + 16, z + 15) + uL0(t0, x + 16, y + 16, z + 17) + uL0(t0, x + 16, y + 17, z + 16) + uL0(t0, x + 17, y + 16, z + 16)) - 4.07312557e-2F*uL0(t0, x + 16, y + 16, z + 16))/(r8*r10 + r9*dampL0(x + 1, y + 1, z + 1));
//    }
//  }
//}

#define SIZEX 800 //On the gpu pointer folding is not allowed with variable size
#define SIZEY 800 //this has to be knowed in compile time
#define SIZEZ 800 //is this bad, for every scale you have to recompile


#define blocksize_x 8
#define blocksize_y 4
#define blocksize_z 32


#define Ddim(arr,x,y,z) arr[(z) + (y)*SIZEZ + (x)*SIZEZ * SIZEY]

const int sizex = SIZEX;
const int sizey = SIZEY;
const int sizez = SIZEZ;

#define uL0(t, x, y, z) u[(t)*x_stride0 + (x)*y_stride0 + (y)*z_stride0 + (z)]
#define dampL0(x, y, z) damp[(x)*y_stride1 + (y)*z_stride1 + (z)]
#define vpL0(x, y, z) vp[(x)*y_stride0 + (y)*z_stride0 + (z)]

__constant__ float r8;
__constant__ float r9;

__constant__ long x_stride0;
__constant__ long y_stride0;
__constant__ long z_stride0;
__constant__ long y_stride1;
__constant__ long z_stride1;
__constant__ long p_rec_stride0;
__constant__ long d_stride0;
__constant__ long p_src_stride0;


__global__ void cuda_section0(const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m,int t0,int t1,int t2,float *vp,float *u,float *damp){
	//printf("section0 gpu\n");

  int z = threadIdx.x + blockIdx.x * blockDim.x + z_m;
  int y = threadIdx.y + blockIdx.y * blockDim.y + y_m;
  int x = threadIdx.z + blockIdx.z * blockDim.z + x_m; 
	if (x <= x_M && y <= y_M && z <= z_M){
      float r10 = 1.0F/(vpL0(x + 16, y + 16, z + 16)*vpL0(x + 16, y + 16, z + 16));
      uL0(t2, x + 16, y + 16, z + 16) = (r9*dampL0(x + 1, y + 1, z + 1)*uL0(t0, x + 16, y + 16, z + 16) + r10*(-r8*(-2.0F*uL0(t0, x + 16, y + 16, z + 16)) - r8*uL0(t1, x + 16, y + 16, z + 16)) + 1.07916777e-8F*(-uL0(t0, x + 8, y + 16, z + 16) - uL0(t0, x + 16, y + 8, z + 16) - uL0(t0, x + 16, y + 16, z + 8) - uL0(t0, x + 16, y + 16, z + 24) - uL0(t0, x + 16, y + 24, z + 16) - uL0(t0, x + 24, y + 16, z + 16)) + 2.25524041e-7F*(uL0(t0, x + 9, y + 16, z + 16) + uL0(t0, x + 16, y + 9, z + 16) + uL0(t0, x + 16, y + 16, z + 9) + uL0(t0, x + 16, y + 16, z + 23) + uL0(t0, x + 16, y + 23, z + 16) + uL0(t0, x + 23, y + 16, z + 16)) + 2.30222458e-6F*(-uL0(t0, x + 10, y + 16, z + 16) - uL0(t0, x + 16, y + 10, z + 16) - uL0(t0, x + 16, y + 16, z + 10) - uL0(t0, x + 16, y + 16, z + 22) - uL0(t0, x + 16, y + 22, z + 16) - uL0(t0, x + 22, y + 16, z + 16)) + 1.54709492e-5F*(uL0(t0, x + 11, y + 16, z + 16) + uL0(t0, x + 16, y + 11, z + 16) + uL0(t0, x + 16, y + 16, z + 11) + uL0(t0, x + 16, y + 16, z + 21) + uL0(t0, x + 16, y + 21, z + 16) + uL0(t0, x + 21, y + 16, z + 16)) + 7.85634138e-5F*(-uL0(t0, x + 12, y + 16, z + 16) - uL0(t0, x + 16, y + 12, z + 16) - uL0(t0, x + 16, y + 16, z + 12) - uL0(t0, x + 16, y + 16, z + 20) - uL0(t0, x + 16, y + 20, z + 16) - uL0(t0, x + 20, y + 16, z + 16)) + 3.35203899e-4F*(uL0(t0, x + 13, y + 16, z + 16) + uL0(t0, x + 16, y + 13, z + 16) + uL0(t0, x + 16, y + 16, z + 13) + uL0(t0, x + 16, y + 16, z + 19) + uL0(t0, x + 16, y + 19, z + 16) + uL0(t0, x + 19, y + 16, z + 16)) + 1.38271608e-3F*(-uL0(t0, x + 14, y + 16, z + 16) - uL0(t0, x + 16, y + 14, z + 16) - uL0(t0, x + 16, y + 16, z + 14) - uL0(t0, x + 16, y + 16, z + 18) - uL0(t0, x + 16, y + 18, z + 16) - uL0(t0, x + 18, y + 16, z + 16)) + 7.90123476e-3F*(uL0(t0, x + 15, y + 16, z + 16) + uL0(t0, x + 16, y + 15, z + 16) + uL0(t0, x + 16, y + 16, z + 15) + uL0(t0, x + 16, y + 16, z + 17) + uL0(t0, x + 16, y + 17, z + 16) + uL0(t0, x + 17, y + 16, z + 16)) - 4.07312557e-2F*uL0(t0, x + 16, y + 16, z + 16))/(r8*r10 + r9*dampL0(x + 1, y + 1, z + 1));
	}

}

__global__ void cuda_test(){
	printf("section0 gpu\n");
}
/*
extern "C" void kernel_section0()
{
	printf("asdasd\n");
	cuda_test<<<2,2>>>();
	hipDeviceSynchronize();
}
*/

extern "C" void kernel_section0( const int x_M, const int x_m, const int y_M, const int y_m, const int z_M, const int z_m, const float dt,int t0,int t1,int t2,float *vp,float *u,float *damp,const long x_stride0,const long y_stride0,const long z_stride0,const long y_stride1,const long z_stride1,const long p_rec_stride0,const long d_stride0,const long p_src_stride0)
{

  float r8 = 1.0F/(dt*dt);
  float r9 = 1.0F/dt;
  hipMemcpyToSymbol(HIP_SYMBOL("r8"), &r8, sizeof(r8));
  hipMemcpyToSymbol(HIP_SYMBOL("r9"), &r9, sizeof(r9));

	hipMemcpyToSymbol(HIP_SYMBOL( "x_stride0"), &x_stride0, sizeof(x_stride0) );
	hipMemcpyToSymbol(HIP_SYMBOL( "y_stride0"), &y_stride0, sizeof(y_stride0) );
	hipMemcpyToSymbol(HIP_SYMBOL( "z_stride0"), &z_stride0, sizeof(z_stride0) );
	hipMemcpyToSymbol(HIP_SYMBOL( "y_stride1"), &y_stride1, sizeof(y_stride1) );
	hipMemcpyToSymbol(HIP_SYMBOL( "z_stride1"), &z_stride1, sizeof(z_stride1) );
	hipMemcpyToSymbol(HIP_SYMBOL( "p_rec_stride0"), &p_rec_stride0, sizeof(p_rec_stride0) );
	hipMemcpyToSymbol(HIP_SYMBOL( "d_stride0"), &d_stride0, sizeof(d_stride0) );
	hipMemcpyToSymbol(HIP_SYMBOL( "p_src_stride0"), &p_src_stride0, sizeof(p_src_stride0) );




	dim3 threads(blocksize_z,blocksize_y,blocksize_x);
	dim3 blocks((SIZEZ-1)/blocksize_z+1,(SIZEY-1)/blocksize_y+1,(SIZEX-1)/blocksize_x+1);
	//printf("blocks dim %d %d %d \n",(SIZEZ-1)/blocksize_z+1,(SIZEY-1)/blocksize_y+1,(SIZEX-1)/blocksize_x+1);
	
	//printf("u pointer gpu: %p\n",u);
	//printf("damp pointer gpu: %p\n",damp);
	//printf("vp pointer gpu: %p\n",vp);
	
	cuda_section0<<<blocks,threads>>>(x_M,x_m,y_M,y_m,z_M,z_m,t0,t1,t2,vp,u,damp);
	hipDeviceSynchronize();
}


extern "C" void cuda_enter_data(float** device_ptr,float* data,size_t size)
{
	hipMalloc(device_ptr, size*sizeof(float));
	hipMemcpy(device_ptr, data,size*sizeof(float), hipMemcpyHostToDevice);
}
extern "C" void cuda_exit_data(float* device_ptr){
	hipFree(device_ptr);
}
extern "C" void cuda_update_data_from(float* device_ptr,float* data,size_t size){
	hipMemcpy(device_ptr, data,size*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(device_ptr);
}



/*

int main()
{
	kernel_section0(10,10,10,10,10,10,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0);



	return 0;
}

*/

