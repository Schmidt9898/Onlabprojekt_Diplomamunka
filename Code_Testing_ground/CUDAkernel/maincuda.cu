#include "hip/hip_runtime.h"
/*******************************************************************************************
*                                                                                          *
*This is an example for making tiling/blocking in CUDA.                                    *
*The calculation purpose is to reach a lot of memory from a single kernel.                 *
*                                                                                          *
*Cpu ram-> gpu vram -> compute -> gpu vram- > cpu ram                                      *
*                                                                                          *
*******************************************************************************************/


//nvcc maincuda.cu -arch=sm_70 -O3 -o maincuda -DL3D

//stopper functions defined here, you may choose not to use them with -D NO_TIME
void Spawn_stopper(char *name);
double Kill_stopper();

#include "stdlib.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define X 8
#define Y 4
#define Z 32


#define SIZEX 800 //On the gpu pointer folding is not allowed with variable size
#define SIZEY 800 //this has to be knowed in compile time
#define SIZEZ 800 //is this bad, for every scale you have to recompile


const int sizex = SIZEX; //If this is not constant, will cause a segfault in runtime with clang-12
const int sizey = SIZEY; //and clang-14 if -D
const int sizez = SIZEZ;// no it will not anymore SIZEXYZ was the answer
                        //BUT if this is not constatn the program will be slower

struct dataobj{void *data;};


#ifdef L1D
  #define Ddim(arr,x,y,z) arr[(z) + (y)*SIZEZ + (x)*SIZEZ * SIZEY]
#elif L3D
  #define Ddim(arr,x,y,z) arr ## _[x][y][z]
#else
  #pragma message "Dimension was not defined use one of L1D or L3D "
#endif

#define KERNEL_WINDOW(out,x,y,z) Ddim(out,x,y,z) += \
                 Ddim(data,x,y,z - 4) + Ddim(data,x,y,z - 3) + Ddim(data,x,y,z - 2) + Ddim(data,x,y,z - 1) +\
                 Ddim(data,x,y,z + 4) + Ddim(data,x,y,z + 3) + Ddim(data,x,y,z + 2) + Ddim(data,x,y,z + 1) +\
                 Ddim(data,x,y - 4,z) + Ddim(data,x,y - 3,z) + Ddim(data,x,y - 2,z) + Ddim(data,x,y - 1,z) +\
                 Ddim(data,x,y + 4,z) + Ddim(data,x,y + 3,z) + Ddim(data,x,y + 2,z) + Ddim(data,x,y + 1,z) +\
                 Ddim(data,x - 4,y,z) + Ddim(data,x - 3,y,z) + Ddim(data,x - 2,y,z) + Ddim(data,x - 1,y,z) +\
                 Ddim(data,x + 4,y,z) + Ddim(data,x + 3,y,z) + Ddim(data,x + 2,y,z) + Ddim(data,x + 1,y,z);




__global__ void init(
#ifdef L1D
  float * __restrict__ data, float * __restrict__ out, float * __restrict__ out2
#else
  float(*__restrict__ data_)[SIZEY][SIZEZ],
  float(*__restrict__ out_)[SIZEY][SIZEZ],
  float(*__restrict__ out2_)[SIZEY][SIZEZ]
#endif
  ) {
  int z = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int x = threadIdx.z + blockIdx.z * blockDim.z;

  if (x < SIZEX && y < SIZEY && z < SIZEZ) {
    Ddim(data,x,y,z) = 1.0f;
    Ddim(out,x,y,z) = 1.0f;
    Ddim(out2,x,y,z) = 1.0f;
  }
}

__global__ void simple(int window_size,
#ifdef L1D
  float * __restrict__ data, float * __restrict__ out
#else
  float(*__restrict__ data_)[SIZEY][SIZEZ],
  float(*__restrict__ out_)[SIZEY][SIZEZ]
#endif
  ) {
  int z = threadIdx.x + blockIdx.x * blockDim.x + window_size;
  int y = threadIdx.y + blockIdx.y * blockDim.y + window_size;
  int x = threadIdx.z + blockIdx.z * blockDim.z + window_size;

  if (x < SIZEX-window_size && y < SIZEY-window_size && z < SIZEZ-window_size) {
    KERNEL_WINDOW(out,x,y,z)
    //printf("xyz %d %d %d  out = %f\n",x,y,z,Ddim(out,x,y,z));
  }
}

int main(int argc, char **argv)
{


printf("Hi this is CUDA tiling test, \n");

#ifdef L1D
  printf("compiled with 1 dimensional arrays.\n");
#elif L3D
  printf("compiled with 3 dimensional arrays.\n");
#else
  #pragma message "Dimension was not defined use one of L1D or L3D "
#endif


const int blocksize_x = X;// if this is not constant the program will be slower
const int blocksize_y = Y;
const int blocksize_z = Z;

printf("block size: %d,%d,%d \n", blocksize_x, blocksize_y, blocksize_z);

printf("thread size needed = %d\n", blocksize_x * blocksize_y * blocksize_z);
//window size
const int window_size = 4;

const size_t meret = sizex * sizey * sizez;
printf("memory size needed: %lu , %f Gb \n", meret * 2, meret * 4 / 1e9f * 2);



Spawn_stopper("offload and memory managment");

//data and offloading OpenMP

//this we will use on the cpu
float * out = (float *)malloc(meret * 4);
float * out2 = (float *)malloc(meret * 4);
float * d_out, *d_out2, *data;
hipMalloc(&d_out, meret*4);
hipMalloc(&d_out2, meret*4);
hipMalloc(&data, meret*4);
Kill_stopper();
{

// pointer folding exact size must be know at compile time
float(*__restrict data_)[SIZEY][SIZEZ] =(float(*__restrict)[SIZEY][SIZEZ])data;
float(*__restrict out_)[SIZEY][SIZEZ] = (float(*__restrict)[SIZEY][SIZEZ])d_out;
float(*__restrict out2_)[SIZEY][SIZEZ] = (float(*__restrict)[SIZEY][SIZEZ])d_out2;

dim3 threads(blocksize_z,blocksize_y,blocksize_x);
dim3 blocks((SIZEZ-1)/blocksize_z+1,(SIZEY-1)/blocksize_y+1,(SIZEX-1)/blocksize_x+1);


printf("blocks dim %d %d %d \n",(SIZEZ-1)/blocksize_z+1,(SIZEY-1)/blocksize_y+1,(SIZEX-1)/blocksize_x+1);

#ifdef L1D
init<<<blocks, threads>>>(data, d_out, d_out2);
#else
init<<<blocks, threads>>>(data_, out_, out2_);
#endif

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
Spawn_stopper("3d computation collapse (3)");
dim3 blocks2((SIZEZ-2*window_size-1)/blocksize_z+1,(SIZEY-2*window_size-1)/blocksize_y+1,(SIZEX-2*window_size-1)/blocksize_x+1);
#ifdef L1D
simple<<<blocks, threads>>>(window_size, data, d_out);
#else
simple<<<blocks, threads>>>(window_size, data_, out_);
#endif
Kill_stopper();

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//Spawn_stopper("3d tiling CUDA");


//Kill_stopper();

///////////////////////////////////////////////////////////////////////////////////////////////////////////////

Spawn_stopper("back to ram");
}
hipMemcpy(out, d_out, meret*4, hipMemcpyDeviceToHost);
hipMemcpy(out2, d_out2, meret*4, hipMemcpyDeviceToHost);
Kill_stopper();






//validation

int good=0;
float(*__restrict data_)[SIZEY][SIZEZ] =(float(*__restrict)[SIZEY][SIZEZ])data;
float(*__restrict out_)[SIZEY][SIZEZ] = (float(*__restrict)[SIZEY][SIZEZ])out;
float(*__restrict out2_)[SIZEY][SIZEZ] = (float(*__restrict)[SIZEY][SIZEZ])out2;

for (int x = window_size; x < sizex - window_size; x++)
{
	for (int y = window_size; y < sizey - window_size; y++)
	{
		for (int z = window_size; z < sizez - window_size; z++)
		{
			if (out_[x][y][z] != 25)
			{
				printf("Validation failed\n");
				printf("out1 %f != 25 \n", out_[x][y][z]);
				good++;
				//goto validationend;
			}
		}
	}
}

/*
for (int i = 0; i < meret; i++) {
  if (out[i]!=out2[i]) {
    printf("Validation failed\n");
  printf("out1 %f != out2 %f\n",out[i],out2[i]);
    good++;
    break;
  }
}
*/
validationend:
if(!good)
  printf("Validation passed\n");

free(out);
free(out2);

return 0;
}

//END of example

//If you dont want to compile this code use -D NO_TIME in compile option.

//This code is for the stopper not part of the example, this was copied and trimed from stopper.h
//Do not change this code
//The example code is up

#ifdef NO_TIME
void Spawn_stopper(char* name){};
double Kill_stopper(){};
#else
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>

typedef struct Stopper;// Stopper();

double Totaltime=0;
int Sum_Stopper=0;
// Stoppers follow LIFO structure
struct Stopper * Stopper_root=NULL; //root of the timers

//get the curent time
//also it does magic because i dont remember what it does actualy
double op_timer_core() {
struct timeval t;
gettimeofday(&t, (struct timezone *)0);
return t.tv_sec + t.tv_usec * 1.0e-6;
}

// proper definition of the stopper
struct Stopper
{
  /* data */
  struct Stopper* parent; //am i a child
  double start,end;       //preaty self explain
  char name[256];          //20 caracter for a name is enought //update it is few week later and 256 because turns out it isn't enought, this took like 3 painfull hours to find out.

};


void Spawn_stopper(char* name)
{
      hipDeviceSynchronize();
      struct Stopper * stopper= (Stopper*)malloc(sizeof(struct Stopper));
      printf("%s ID:%d started->\n",name,Sum_Stopper);
      strcpy(stopper->name,name);
      stopper->start=op_timer_core();
      stopper->parent=Stopper_root;
      Stopper_root=stopper;
      Sum_Stopper++;
}
double Kill_stopper()
{
  hipDeviceSynchronize();
  double ret=0;
  if(Stopper_root==NULL)
      return ret;
  Stopper_root->end=op_timer_core();
  ret=Stopper_root->end-=Stopper_root->start;
  printf("%s took: %f sec.\n",Stopper_root->name,Stopper_root->end);
  struct Stopper* t=Stopper_root;//free the stopper
  Stopper_root=Stopper_root->parent;
  free(t);
  return ret;

}
#endif